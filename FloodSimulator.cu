#include "FloodSimulator.cuh"
#include "VirtualPipelineModel.cuh"

void FloodSimulator::SetDeltaT(double newDeltaT)
{
	this->deltaT = newDeltaT;
}

double FloodSimulator::GetDeltaT() const
{
	return this->deltaT;
}

void FloodSimulator::SetPipeLength(double newPipeLength)
{
	this->pipeLength = newPipeLength;
}

double FloodSimulator::GetPipeLength() const
{
	return this->pipeLength;
}

void FloodSimulator::SetGravity(double newGravity)
{
	this->gravity = newGravity;
}

double FloodSimulator::GetGravity() const
{
	return this->gravity;
}

void FloodSimulator::SetKe(double newKe)
{
	this->Ke = newKe;
}

double FloodSimulator::GetKe() const
{
	return this->Ke;
}

void FloodSimulator::InitDevice()
{
	hipMalloc((void**)&gTerrainHeight, sizeof(double) * sizeX * sizeY);
	hipMalloc((void**)&gBuildingHeight, sizeof(double) * sizeX * sizeY);
	hipMalloc((void**)&gDamHeight, sizeof(double) * sizeX * sizeY);
	hipMalloc((void**)&gSurfaceHeight, sizeof(double) * sizeX * sizeY);
	hipMalloc((void**)&gWaterHeight, sizeof(double) * sizeX * sizeY);
	hipMalloc((void**)&gRainfallRate, sizeof(double) * numRainfallLayer * sizeX * sizeY);
	hipMalloc((void**)&gDrainRate, sizeof(double) * sizeX * sizeY);
	hipMalloc((void**)&gRiverInflow, sizeof(Vec3) * sizeX * sizeY);
	hipMalloc((void**)&gWaterVelocity, sizeof(Vec2) * sizeX * sizeY);
	hipMalloc((void**)&gFlowField, sizeof(FlowField) * sizeX * sizeY);
	hipMalloc((void**)&gNewFlowField, sizeof(FlowField) * sizeX * sizeY);
}

void FloodSimulator::SendAllDataToDevice()
{
	hipMemcpy(gTerrainHeight, &terrainHeight[0], sizeof(double) * sizeX * sizeY, hipMemcpyHostToDevice);
	hipMemcpy(gBuildingHeight, &buildingHeight[0], sizeof(double) * sizeX * sizeY, hipMemcpyHostToDevice);
	hipMemcpy(gDamHeight, &damHeight[0], sizeof(double) * sizeX * sizeY, hipMemcpyHostToDevice);
	hipMemcpy(gSurfaceHeight, &surfaceHeight[0], sizeof(double) * sizeX * sizeY, hipMemcpyHostToDevice);
	hipMemcpy(gWaterHeight, &waterHeight[0], sizeof(double) * sizeX * sizeY, hipMemcpyHostToDevice);
	hipMemcpy(gRainfallRate, &rainfallRate[0], sizeof(double) * numRainfallLayer * sizeX * sizeY, hipMemcpyHostToDevice);
	hipMemcpy(gDrainRate, &drainRate[0], sizeof(double) * sizeX * sizeY, hipMemcpyHostToDevice);
	hipMemcpy(gRiverInflow, &riverInflow[0], sizeof(Vec3) * sizeX * sizeY, hipMemcpyHostToDevice);
	hipMemcpy(gWaterVelocity, &waterVelocity[0], sizeof(Vec2) * sizeX * sizeY, hipMemcpyHostToDevice);
	hipMemcpy(gFlowField, &flowField[0], sizeof(FlowField) * sizeX * sizeY, hipMemcpyHostToDevice);
	hipMemcpy(gNewFlowField, &newFlowField[0], sizeof(FlowField) * sizeX * sizeY, hipMemcpyHostToDevice);
}

void FloodSimulator::PreparationForSimulaion()
{
	dim3 dimBlock(32, 16);
	dim3 dimGrid((sizeX - 1) / dimBlock.x + 1, (sizeY - 1) / dimBlock.y + 1);

	InitFlowFields << < dimGrid, dimBlock >> > (gFlowField, gNewFlowField, sizeX, sizeY);
	InitVelocity << < dimGrid, dimBlock >> > (gWaterVelocity, sizeX, sizeY);
	UpdateSurfaceHeight << < dimGrid, dimBlock >> > (gTerrainHeight, gBuildingHeight, gSurfaceHeight, sizeX, sizeY);
}

void FloodSimulator::RunSimulation(const unsigned int step)
{
	dim3 dimBlock(32, 16);
	dim3 dimGrid((sizeX - 1) / dimBlock.x + 1, (sizeY - 1) / dimBlock.y + 1);

	WaterIncrementByRainfall << < dimGrid, dimBlock >> > (gWaterHeight, gRainfallRate, sizeX, sizeY, deltaT, numRainfallLayer, step, 3000);
	UpdateOutputFlowField << < dimGrid, dimBlock >> > (gFlowField, gNewFlowField, gSurfaceHeight, gWaterHeight, sizeX, sizeY, deltaT, pipeLength, gravity);
	UpdateNewFlowField << < dimGrid, dimBlock >> > (gFlowField, gNewFlowField, sizeX, sizeY);
	UpdateWaterVelocityAndHeight << < dimGrid, dimBlock >> > (gWaterHeight, gWaterVelocity, gFlowField, sizeX, sizeY, deltaT, pipeLength);
	//Evaporation << < dimGrid, dimBlock >> > (gWaterHeight, sizeX, sizeY, Ke, deltaT);

}

void FloodSimulator::GetResultFromDevice()
{
	hipMemcpy(&waterHeight[0], gWaterHeight, sizeof(double) * sizeX * sizeY, hipMemcpyDeviceToHost);
	hipMemcpy(&waterVelocity[0], gWaterVelocity, sizeof(Vec2) * sizeX * sizeY, hipMemcpyDeviceToHost);
}

void FloodSimulator::FreeAllData()
{
	hipFree(gTerrainHeight);
	hipFree(gBuildingHeight);
	hipFree(gDamHeight);
	hipFree(gSurfaceHeight);
	hipFree(gWaterHeight);
	hipFree(gRainfallRate);
	hipFree(gDrainRate);
	hipFree(gRiverInflow);
	hipFree(gWaterVelocity);
	hipFree(gFlowField);
	hipFree(gNewFlowField);
}